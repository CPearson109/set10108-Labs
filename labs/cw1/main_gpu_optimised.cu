#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <cstring>

// Function to read the file content
std::vector<char> read_file(const char* filename)
{
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error: Could not open the file " << filename << std::endl;
        return {};
    }

    file.seekg(0, std::ios::end);
    std::streamsize fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    std::vector<char> buffer(fileSize);
    if (!file.read(buffer.data(), fileSize)) {
        std::cerr << "Error: Could not read the file content." << std::endl;
        return {};
    }

    file.close();
    std::transform(buffer.begin(), buffer.end(), buffer.begin(), [](char c) { return std::tolower(c); });

    return buffer;
}

// CUDA kernel to count occurrences of words
__global__ void count_token_occurrences(const char* data, int data_size, const char* const* words, int* word_lengths, int num_words, int* counts)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= data_size) return;

    for (int w = 0; w < num_words; ++w)
    {
        int token_length = word_lengths[w];
        if (idx + token_length > data_size) continue;

        bool match = true;
        for (int i = 0; i < token_length; ++i)
        {
            if (data[idx + i] != words[w][i])
            {
                match = false;
                break;
            }
        }
        if (!match) continue;

        int iPrefix = idx - 1;
        if (iPrefix >= 0 && data[iPrefix] >= 'a' && data[iPrefix] <= 'z') continue;

        int iSuffix = idx + token_length;
        if (iSuffix < data_size && data[iSuffix] >= 'a' && data[iSuffix] <= 'z') continue;

        atomicAdd(&counts[w], 1);
    }
}

int main()
{
    // Example file path
    const char* filepath = "dataset/shakespeare.txt";

    // Read file data
    std::vector<char> file_data = read_file(filepath);
    if (file_data.empty()) return -1;

    // Copy file data to the GPU
    size_t data_size = file_data.size();
    char* d_data;
    hipMalloc((void**)&d_data, data_size);
    hipMemcpy(d_data, file_data.data(), data_size, hipMemcpyHostToDevice);

    // Example word list
    const char* words[] = { "sword", "fire", "death", "love", "hate", "the", "man", "woman" };
    int num_words = sizeof(words) / sizeof(words[0]);

    // Allocate memory for word pointers and lengths on the GPU
    char** d_words;
    int* d_word_lengths;
    hipMalloc((void**)&d_words, num_words * sizeof(char*));
    hipMalloc((void**)&d_word_lengths, num_words * sizeof(int));

    // Allocate memory for the occurrence counts on the GPU
    int* d_counts;
    hipMalloc((void**)&d_counts, num_words * sizeof(int));
    hipMemset(d_counts, 0, num_words * sizeof(int));

    // Prepare word lengths and copy words to the GPU
    std::vector<int> word_lengths(num_words);
    for (int i = 0; i < num_words; ++i)
    {
        word_lengths[i] = strlen(words[i]);
        char* d_word;
        hipMalloc((void**)&d_word, word_lengths[i]);
        hipMemcpy(d_word, words[i], word_lengths[i], hipMemcpyHostToDevice);
        hipMemcpy(&d_words[i], &d_word, sizeof(char*), hipMemcpyHostToDevice);
    }
    hipMemcpy(d_word_lengths, word_lengths.data(), num_words * sizeof(int), hipMemcpyHostToDevice);

    // Create CUDA events for total timing
    hipEvent_t total_start, total_stop;
    hipEventCreate(&total_start);
    hipEventCreate(&total_stop);

    // Record the start event for total time
    hipEventRecord(total_start);

    // Loop through each word, time individually, and launch kernel
    for (int w = 0; w < num_words; ++w)
    {
        // Create CUDA events for timing each word
        hipEvent_t word_start, word_stop;
        hipEventCreate(&word_start);
        hipEventCreate(&word_stop);

        // Record the start event for each word
        hipEventRecord(word_start);

        // Launch kernel for the current word
        int threadsPerBlock = 256;
        int blocksPerGrid = (data_size + threadsPerBlock - 1) / threadsPerBlock;
        count_token_occurrences << <blocksPerGrid, threadsPerBlock >> > (d_data, data_size, d_words, d_word_lengths, num_words, d_counts);

        // Record the stop event for each word
        hipEventRecord(word_stop);
        hipEventSynchronize(word_stop);

        // Calculate the elapsed time for each word
        float word_time_ms = 0;
        hipEventElapsedTime(&word_time_ms, word_start, word_stop);

        // Copy counts back to host for this word
        int occurrences = 0;
        hipMemcpy(&occurrences, &d_counts[w], sizeof(int), hipMemcpyDeviceToHost);

        // Print the result for the current word
        std::cout << "Found " << occurrences << " occurrences of word: " << words[w] << " in " << word_time_ms << " ms." << std::endl;

        // Destroy the word events
        hipEventDestroy(word_start);
        hipEventDestroy(word_stop);
    }

    // Record the stop event for total time
    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);

    // Calculate the total elapsed time
    float total_time_ms = 0;
    hipEventElapsedTime(&total_time_ms, total_start, total_stop);

    std::cout << "Total time to find all occurrences: " << total_time_ms << " ms." << std::endl;

    // Free device memory
    for (int i = 0; i < num_words; ++i)
    {
        char* d_word;
        hipMemcpy(&d_word, &d_words[i], sizeof(char*), hipMemcpyHostToDevice);
        hipFree(d_word);
    }
    hipFree(d_words);
    hipFree(d_word_lengths);
    hipFree(d_counts);
    hipFree(d_data);

    // Destroy the total events
    hipEventDestroy(total_start);
    hipEventDestroy(total_stop);

    return 0;
}
